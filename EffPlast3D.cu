#include "hip/hip_runtime.h"
#include "EffPlast3D.h"

__global__ void ComputeDisp(double* Ux, double* Uy, double* Uz, double* Vx, double* Vy, double* Vz,
  const double* const P,
  const double* const tauXX, const double* const tauYY, const double* const tauZZ,
  const double* const tauXY, const double* const tauXZ, const double* const tauYZ,
  const double* const pa,
  const long int nX, const long int nY, const long int nZ)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  const double dX = pa[0], dY = pa[1], dZ = pa[2];
  const double dT = pa[3];
  const double rho = pa[6];
  const double dampX = pa[7], dampY = pa[8], dampZ = pa[9];

  // motion equation
  if (i > 0 && i < nX && j > 0 && j < nY - 1 && k > 0 && k < nZ - 1) {
    Vx[k * (nX + 1) * nY + j * (nX + 1) + i] = Vx[k * (nX + 1) * nY + j * (nX + 1) + i] * (1.0 - dT * dampX) + (dT / rho) * ((
      -P[k * nX * nY + j * nX + i] + P[k * nX * nY + j * nX + i - 1] + tauXX[k * nX * nY + j * nX + i] - tauXX[k * nX * nY + j * nX + i - 1]
      ) / dX +
      (
        tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i - 1] - tauXY[k * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i - 1]
        ) / dY +
      (
        tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i - 1] - tauXZ[(k - 1) * (nX - 1) * nY + j * (nX - 1) + i - 1]
        ) / dZ
      );
  }
  if (i > 0 && i < nX - 1 && j > 0 && j < nY && k > 0 && k < nZ - 1) {
    Vy[k * nX * (nY + 1) + j * nX + i] = Vy[k * nX * (nY + 1) + j * nX + i] * (1.0 - dT * dampY) + (dT / rho) * ((    // why dT * dampY ?
      -P[k * nX * nY + j * nX + i] + P[k * nX * nY + (j - 1) * nX + i] + tauYY[k * nX * nY + j * nX + i] - tauYY[k * nX * nY + (j - 1) * nX + i]
      ) / dY +
      (
        tauXY[k * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i] - tauXY[k * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i - 1]
        ) / dX +
      (
        tauYZ[k * nX * (nY - 1) + (j - 1) * nX + i] - tauYZ[(k - 1) * nX * (nY - 1) + (j - 1) * nX + i]
        ) / dZ
      );
  }
  if (i > 0 && i < nX - 1 && j > 0 && j < nY - 1 && k > 0 && k < nZ) {
    Vz[k * nX * nY + j * nX + i] = Vz[k * nX * nY + j * nX + i] * (1.0 - dT * dampZ) + (dT / rho) * ((
      -P[k * nX * nY + j * nX + i] + P[(k - 1) * nX * nY + j * nX + i] + tauZZ[k * nX * nY + j * nX + i] - tauZZ[(k - 1) * nX * nY + j * nX + i]
      ) / dZ +
      (
        tauXZ[(k - 1) * (nX - 1) * nY + j * (nX - 1) + i] - tauXZ[(k - 1) * (nX - 1) * nY + j * (nX - 1) + i - 1]
        ) / dX +
      (
        tauYZ[(k - 1) * nX * (nY - 1) + j * nX + i] - tauYZ[(k - 1) * nX * (nY - 1) + (j - 1) * nX + i]
        ) / dY
      );
  }

  Ux[k * (nX + 1) * nY + j * (nX + 1) + i] = Ux[k * (nX + 1) * nY + j * (nX + 1) + i] + Vx[k * (nX + 1) * nY + j * (nX + 1) + i] * dT;
  Uy[k * nX * (nY + 1) + j * nX + i] = Uy[k * nX * (nY + 1) + j * nX + i] + Vy[k * nX * (nY + 1) + j * nX + i] * dT;
  Uz[k * nX * nY + j * nX + i] = Uz[k * nX * nY + j * nX + i] + Vz[k * nX * nY + j * nX + i] * dT;
}

__global__ void ComputeStress(const double* const Ux, const double* const Uy, const double* Uz,
  const double* const K, const double* const G,
  const double* const P0, double* P,
  double* tauXX, double* tauYY, double* tauZZ,
  double* tauXY, double* tauXZ, double* tauYZ,
  const double* const pa,
  const long int nX, const long int nY, const long int nZ)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  const double dX = pa[0], dY = pa[1], dZ = pa[2];
  // const double dT = pa[2];
  const double rad = pa[11];
  const double N = pa[12];

  // constitutive equation - Hooke's law
  P[k * nX * nY + j * nX + i] = P0[k * nX * nY + j * nX + i] - K[k * nX * nY + j * nX + i] * (
    (Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i]) / dX +     // divU
    (Uy[k * nX * (nY + 1) + (j + 1) * nX + i] - Uy[k * nX * (nY + 1) + j * nX + i]) / dY +
    (Uz[(k + 1) * nX * nY + j * nX + i] - Uz[k * nX * nY + j * nX + i]) / dZ
    );

  /*P[j * nX + i] = P[j * nX + i] - G[j * nX + i] * ( // incompressibility
  (Ux[j * (nX + 1) + i + 1] - Ux[j * (nX + 1) + i]) / dX + (Uy[(j + 1) * nX + i] - Uy[j * nX + i]) / dY    // divU
  ) * dT / nX;*/

  tauXX[k * nX * nY + j * nX + i] = 2.0 * G[k * nX * nY + j * nX + i] * (
    (Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i]) / dX -    // dUx/dx
      (
        (Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i]) / dX +
        (Uy[k * nX * (nY + 1) + (j + 1) * nX + i] - Uy[k * nX * (nY + 1) + j * nX + i]) / dY +
        (Uz[(k + 1) * nX * nY + j * nX + i] - Uz[k * nX * nY + j * nX + i]) / dZ
      ) / 3.0    // divU / 3.0
    );
  tauYY[k * nX * nY + j * nX + i] = 2.0 * G[k * nX * nY + j * nX + i] * (
    (Uy[k * nX * (nY + 1) + (j + 1) * nX + i] - Uy[k * nX * (nY + 1) + j * nX + i]) / dY -    // dUy/dy
      (
        (Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i]) / dX +
        (Uy[k * nX * (nY + 1) + (j + 1) * nX + i] - Uy[k * nX * (nY + 1) + j * nX + i]) / dY +
        (Uz[(k + 1) * nX * nY + j * nX + i] - Uz[k * nX * nY + j * nX + i]) / dZ
      ) / 3.0    // divU / 3.0
    );
  tauZZ[k * nX * nY + j * nX + i] = 2.0 * G[k * nX * nY + j * nX + i] * (
    (Uz[(k + 1) * nX * nY + j * nX + i] - Uz[k * nX * nY + j * nX + i]) / dZ -    // dUz/dz
      (
        (Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i]) / dX +
        (Uy[k * nX * (nY + 1) + (j + 1) * nX + i] - Uy[k * nX * (nY + 1) + j * nX + i]) / dY +
        (Uz[(k + 1) * nX * nY + j * nX + i] - Uz[k * nX * nY + j * nX + i]) / dZ
      ) / 3.0    // divU / 3.0
    );

  if (i < nX - 1 && j < nY - 1) {
    tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] = 0.25 * (
        G[k * nX * nY + j * nX + i] + G[k * nX * nY + j * nX + i + 1] + G[k * nX * nY + (j + 1) * nX + i] + G[k * nX * nY + (j + 1) * nX + i + 1]
      ) * (
        (Ux[k * (nX + 1) * nY + (j + 1) * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1]) / dY + 
        (Uy[k * nX * (nY + 1) + (j + 1) * nX + i + 1] - Uy[k * nX * (nY + 1) + (j + 1) * nX + i]) / dX    // dUx/dy + dUy/dx
      );
  }
  if (i < nX - 1 && k < nZ - 1) {
    tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i] = 0.25 * (
        G[k * nX * nY + j * nX + i] + G[k * nX * nY + j * nX + i + 1] + G[(k + 1) * nX * nY + j * nX + i] + G[(k + 1) * nX * nY + j * nX + i + 1]
      ) * (
        (Ux[(k + 1) * (nX + 1) * nY + j * (nX + 1) + i + 1] - Ux[k * (nX + 1) * nY + j * (nX + 1) + i + 1]) / dZ +
        (Uz[(k + 1) * nX * nY + j * nX + i + 1] - Uz[(k + 1) * nX * nY + j * nX + i]) / dX    // dUx/dz + dUz/dx
      );
  }
  if (j < nY - 1 && k < nZ - 1) {
    tauYZ[k * nX * (nY - 1) + j * nX + i] = 0.25 * (
        G[k * nX * nY + j * nX + i] + G[k * nX * nY + (j + 1) * nX + i] + G[(k + 1) * nX * nY + j * nX + i] + G[(k + 1) * nX * nY + (j + 1) * nX + i]
      ) * (
        (Uy[(k + 1) * nX * (nY + 1) + (j + 1) * nX + i] - Uy[k * nX * (nY + 1) + (j + 1) * nX + i]) / dZ +
        (Uz[(k + 1) * nX * nY + (j + 1) * nX + i] - Uz[(k + 1) * nX * nY + j * nX + i]) / dY    // dUy/dz + dUz/dy
      );
  }

  for (int a = 0; a < N; a++) {
    for (int b = 0; b < N; b++) {
      for (int c = 0; c < N; c++) {
        if (sqrt((-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) *
          (-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) +
          (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) *
          (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) +
          (-0.5 * dZ * (nZ - 1) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c) * 
          (-0.5 * dZ * (nZ - 1) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c)) < rad) {
          P[k * nX * nY + j * nX + i] = 0.0;
          tauXX[k * nX * nY + j * nX + i] = 0.0;
          tauYY[k * nX * nY + j * nX + i] = 0.0;
          tauZZ[k * nX * nY + j * nX + i] = 0.0;
        }

        if (i < nX - 1 && j < nY - 1) {
          if (sqrt((-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) *
            (-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) +
            (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) *
            (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) +
            (-0.5 * dZ * (nZ - 1) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c) * 
            (-0.5 * dZ * (nZ - 1) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c)) < rad) {
            tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] = 0.0;
          }
        }
        if (i < nX - 1 && k < nZ - 1) {
          if (sqrt((-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) *
            (-0.5 * dX * (nX - 2) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) +
            (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) *
            (-0.5 * dY * (nY - 1) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) +
            (-0.5 * dZ * (nZ - 2) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c) * 
            (-0.5 * dZ * (nZ - 2) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c)) < rad) {
            tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i] = 0.0;
          }
        }
        if (j < nY - 1 && k < nZ - 1) {
          if (sqrt((-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) *
            (-0.5 * dX * (nX - 1) + dX * i - 0.5 * dX * (nX - 1) * (1.0 - 1.0 / N) + (dX * (nX - 1) / N) * a) +
            (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) *
            (-0.5 * dY * (nY - 2) + dY * j - 0.5 * dY * (nY - 1) * (1.0 - 1.0 / N) + (dY * (nY - 1) / N) * b) +
            (-0.5 * dZ * (nZ - 2) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c) * 
            (-0.5 * dZ * (nZ - 2) + dZ * k - 0.5 * dZ * (nZ - 1) * (1.0 - 1.0 / N) + (dZ * (nZ - 1) / N) * c)) < rad) {
            tauYZ[k * nX * (nY - 1) + j * nX + i] = 0.0;
          }
        }
      } // for(c)
    } // for(b)
  } // for(a)
}

__global__ void ComputeJ2(double* tauXX, double* tauYY, double* tauZZ,
  double* tauXY, double* tauXZ, double* tauYZ,
  double* const tauXYav, double* const tauXZav, double* const tauYZav,
  double* const J2, double* const J2XY, double* const J2XZ, double* const J2YZ,
  const long int nX, const long int nY, const long int nZ)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  // tauIJ for plasticity
  if (i > 0 && i < nX - 1 && j > 0 && j < nY - 1) {
    tauXYav[k * nX * nY + j * nX + i] = 0.25 * (
      tauXY[k * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i - 1] + tauXY[k * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i] + 
      tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i - 1] + tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i]
    );
  }
  if (i > 0 && i < nX - 1 && k > 0 && k < nZ - 1) {
    tauXZav[k * nX * nY + j * nX + i] = 0.25 * (
      tauXZ[(k - 1) * (nX - 1) * nY + j * (nX - 1) + i - 1] + tauXZ[(k - 1) * (nX - 1) * nY + j * (nX - 1) + i] +
      tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i - 1] + tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i]
    );
  }
  if (j > 0 && j < nY - 1 && k > 0 && k < nZ - 1) {
    tauYZav[k * nX * nY + j * nX + i] = 0.25 * (
      tauYZ[(k - 1) * nX * (nY - 1) + (j - 1) * nX + i] + tauYZ[(k - 1) * nX * (nY - 1) + j * nX + i] +
      tauYZ[k * nX * (nY - 1) + (j - 1) * nX + i] + tauYZ[k * nX * (nY - 1) + j * nX + i]
    );
  }
  J2[k * nX * nY + j * nX + i] = sqrt(
    tauXX[k * nX * nY + j * nX + i] * tauXX[k * nX * nY + j * nX + i] + 
    tauYY[k * nX * nY + j * nX + i] * tauYY[k * nX * nY + j * nX + i] +
    tauZZ[k * nX * nY + j * nX + i] * tauZZ[k * nX * nY + j * nX + i] +
    2.0 * (
      tauXYav[k * nX * nY + j * nX + i] * tauXYav[k * nX * nY + j * nX + i] +
      tauXZav[k * nX * nY + j * nX + i] * tauXZav[k * nX * nY + j * nX + i] +
      tauYZav[k * nX * nY + j * nX + i] * tauYZav[k * nX * nY + j * nX + i]
    )
  );
  if (i < nX - 1 && j < nY - 1 && k > 0 && k < nZ - 1) {
    J2XY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] = sqrt(
      pow(0.25 * (tauXX[k * nX * nY + (j + 1) * nX + i + 1] + tauXX[k * nX * nY + (j + 1) * nX + i] + tauXX[k * nX * nY + j * nX + i + 1] + tauXX[k * nX * nY + j * nX + i]), 2.0) +
      pow(0.25 * (tauYY[k * nX * nY + (j + 1) * nX + i + 1] + tauYY[k * nX * nY + (j + 1) * nX + i] + tauYY[k * nX * nY + j * nX + i + 1] + tauYY[k * nX * nY + j * nX + i]), 2.0) +
      pow(0.25 * (tauZZ[k * nX * nY + (j + 1) * nX + i + 1] + tauZZ[k * nX * nY + (j + 1) * nX + i] + tauZZ[k * nX * nY + j * nX + i + 1] + tauZZ[k * nX * nY + j * nX + i]), 2.0) +
      2.0 * (
        pow(tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i], 2.0) +
        pow(0.25 * (
          tauXZ[(k - 1) * (nX - 1) * nY + j * (nX - 1) + i] + tauXZ[(k - 1) * (nX - 1) * nY + (j + 1) * (nX - 1) + i] + 
          tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i] + tauXZ[k * (nX - 1) * nY + (j + 1) * (nX - 1) + i]
        ), 2.0) +
        pow(0.25 * (
          tauYZ[(k - 1) * nX * (nY - 1) + j * nX + i] + tauYZ[(k - 1) * nX * (nY - 1) + j * nX + i + 1] +
          tauYZ[k * nX * (nY - 1) + j * nX + i] + tauYZ[k * nX * (nY - 1) + j * nX + i + 1]
        ), 2.0)
      )
    ); // sqrt
  }
  if (i < nX - 1 && j > 0 && j < nY - 1 && k < nZ - 1) {
    J2XZ[k * (nX - 1) * nY + j * (nX - 1) + i] = sqrt(
      pow(0.25 * (
        tauXX[k * nX * nY + j * nX + i + 1] + tauXX[k * nX * nY + j * nX + i] + tauXX[(k + 1) * nX * nY + j * nX + i + 1] + tauXX[(k + 1) * nX * nY + j * nX + i]
        ), 2.0) +
      pow(0.25 * (
        tauYY[k * nX * nY + j * nX + i + 1] + tauYY[k * nX * nY + j * nX + i] + tauYY[(k + 1) * nX * nY + j * nX + i + 1] + tauYY[(k + 1) * nX * nY + j * nX + i]
        ), 2.0) +
      pow(0.25 * (
        tauZZ[k * nX * nY + j * nX + i + 1] + tauZZ[k * nX * nY + j * nX + i] + tauZZ[(k + 1) * nX * nY + j * nX + i + 1] + tauZZ[(k + 1) * nX * nY + j * nX + i]
        ), 2.0) +
      2.0 * (
        pow(tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i], 2.0) +
        pow(0.25 * (
          tauXY[k * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i] + tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] +
          tauXY[(k + 1) * (nX - 1) * (nY - 1) + (j - 1) * (nX - 1) + i] + tauXY[(k + 1) * (nX - 1) * (nY - 1) + j * (nX - 1) + i]
        ), 2.0) +
        pow(0.25 * (
          tauYZ[k * nX * (nY - 1) + (j - 1) * nX + i] + tauYZ[k * nX * (nY - 1) + (j - 1) * nX + i + 1] +
          tauYZ[k * nX * (nY - 1) + j * nX + i] + tauYZ[k * nX * (nY - 1) + j * nX + i + 1]
        ), 2.0)
      )
    ); // sqrt
  }
  if (i > 0 && i < nX - 1 && j < nY - 1 && k < nZ - 1) {
    J2YZ[k * nX * (nY - 1) + j * nX + i] = sqrt(
      pow(0.25 * (
        tauXX[(k + 1) * nX * nY + j * nX + i] + tauXX[k * nX * nY + j * nX + i] + tauXX[(k + 1) * nX * nY + (j + 1) * nX + i] + tauXX[k * nX * nY + (j + 1) * nX + i]
        ), 2.0) +
      pow(0.25 * (
        tauYY[(k + 1) * nX * nY + j * nX + i] + tauYY[k * nX * nY + j * nX + i] + tauYY[(k + 1) * nX * nY + (j + 1) * nX + i] + tauYY[k * nX * nY + (j + 1) * nX + i]
        ), 2.0) +
      pow(0.25 * (
        tauZZ[(k + 1) * nX * nY + j * nX + i] + tauZZ[k * nX * nY + j * nX + i] + tauZZ[(k + 1) * nX * nY + (j + 1) * nX + i] + tauZZ[k * nX * nY + (j + 1) * nX + i]
        ), 2.0) +
      2.0 * (
        pow(tauYZ[k * nX * (nY - 1) + j * nX + i], 2.0) +
        pow(0.25 * (
          tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i - 1] + tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] +
          tauXY[(k + 1) * (nX - 1) * (nY - 1) + j * (nX - 1) + i - 1] + tauXY[(k + 1) * (nX - 1) * (nY - 1) + j * (nX - 1) + i]
        ), 2.0) +
        pow(0.25 * (
          tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i - 1] + tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i] +
          tauXZ[k * (nX - 1) * nY + (j + 1) * (nX - 1) + i - 1] + tauXZ[k * (nX - 1) * nY + (j + 1) * (nX - 1) + i]
        ), 2.0)
      )
    ); // sqrt
  }
}

__global__ void ComputePlasticity(double* tauXX, double* tauYY, double* tauZZ,
  double* tauXY, double* tauXZ, double* tauYZ,
  double* const tauXYav, double* const tauXZav, double* const tauYZav,
  double* const J2, double* const J2XY, double* const J2XZ, double* const J2YZ,
  const long int nX, const long int nY, const long int nZ)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  int k = blockIdx.z * blockDim.z + threadIdx.z;

  // plasticity
  if (J2[k * nX * nY + j * nX + i] > coh_cuda) {
    tauXX[k * nX * nY + j * nX + i] *= coh_cuda / J2[k * nX * nY + j * nX + i];
    tauYY[k * nX * nY + j * nX + i] *= coh_cuda / J2[k * nX * nY + j * nX + i];
    tauZZ[k * nX * nY + j * nX + i] *= coh_cuda / J2[k * nX * nY + j * nX + i];
    tauXYav[k * nX * nY + j * nX + i] *= coh_cuda / J2[k * nX * nY + j * nX + i];
    tauXZav[k * nX * nY + j * nX + i] *= coh_cuda / J2[k * nX * nY + j * nX + i];
    tauYZav[k * nX * nY + j * nX + i] *= coh_cuda / J2[k * nX * nY + j * nX + i];
  }

  if (i < nX - 1 && j < nY - 1) {
    if (J2XY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] > coh_cuda) {
      tauXY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i] *= coh_cuda / J2XY[k * (nX - 1) * (nY - 1) + j * (nX - 1) + i];
    }
  }
  if (i < nX - 1 && k < nZ - 1) {
    if (J2XZ[k * (nX - 1) * nY + j * (nX - 1) + i] > coh_cuda) {
      tauXZ[k * (nX - 1) * nY + j * (nX - 1) + i] *= coh_cuda / J2XZ[k * (nX - 1) * nY + j * (nX - 1) + i];
    }
  }
  if (j < nY - 1 && k < nZ - 1) {
    if (J2YZ[k * nX * (nY - 1) + j * nX + i] > coh_cuda) {
      tauYZ[k * nX * (nY - 1) + j * nX + i] *= coh_cuda / J2YZ[k * nX * (nY - 1) + j * nX + i];
    }
  }
  
  // recalculation J2 for correct visualization only
  J2[k * nX * nY + j * nX + i] = sqrt(
    tauXX[k * nX * nY + j * nX + i] * tauXX[k * nX * nY + j * nX + i] + 
    tauYY[k * nX * nY + j * nX + i] * tauYY[k * nX * nY + j * nX + i] +
    tauZZ[k * nX * nY + j * nX + i] * tauZZ[k * nX * nY + j * nX + i] +
    2.0 * (
      tauXYav[k * nX * nY + j * nX + i] * tauXYav[k * nX * nY + j * nX + i] +
      tauXZav[k * nX * nY + j * nX + i] * tauXZav[k * nX * nY + j * nX + i] +
      tauYZav[k * nX * nY + j * nX + i] * tauYZav[k * nX * nY + j * nX + i]
      )
  );
}

double EffPlast3D::ComputeEffModuli(const double initLoadValue, [[deprecated]] const double loadValue, 
  const unsigned int nTimeSteps, const std::array<double, 6>& loadType)
{
  if (nPores <= 0) {
    throw std::invalid_argument("Error! The number of pores must be positive!\n");
  }

  const auto start = std::chrono::system_clock::now();
  nTimeSteps_ = nTimeSteps;
  loadType_ = loadType;

  std::array<double, 6> sphericalLoadType{
    (loadType_[0] + loadType_[1] + loadType_[2]) / 3.0,
    (loadType_[0] + loadType_[1] + loadType_[2]) / 3.0,
    (loadType_[0] + loadType_[1] + loadType_[2]) / 3.0,
    0.0, 0.0, 0.0
  };
  //std::array<double, 6> deviatoricLoadType{loadType_[0] - sphericalLoadType[0], loadType_[1] - sphericalLoadType[1], loadType_[2]};

  printCalculationType();

  ComputeEffParams(0, initLoadValue, loadType_, nTimeSteps_);
  if (NL == 1) {
    calcBulkModuli_PureElast();
  }
  else {
    ComputeEffParams(1, initLoadValue * incPercent, sphericalLoadType, 1);
    calcBulkModuli_ElastPlast();
  }

  /*if (NL == 3) {
    ComputeEffParams(2, initLoadValue * incPercent, deviatoricLoadType, 1);
    calcShearModulus();
  }

  printEffectiveModuli();
  printWarnings();
  */

  /* OUTPUT DATA WRITING */
  SaveSlice(P_cpu, P_cuda, nX, nY, nZ, nZ / 2, "data/PcXY_" + std::to_string(8 * NGRID) + "_.dat");
  SaveSlice(tauXX_cpu, tauXX_cuda, nX, nY, nZ, nZ / 2, "data/tauXXc_" + std::to_string(8 * NGRID) + "_.dat");
  SaveSlice(tauXZ_cpu, tauXZ_cuda, nX - 1, nY, nZ - 1, nZ / 2, "data/tauXZcXY_" + std::to_string(8 * NGRID) + "_.dat");
  if (NL > 1) {
    SaveSlice(J2_cpu, J2_cuda, nX, nY, nZ, nZ / 2, "data/J2cXY_" + std::to_string(8 * NGRID) + "_.dat");
    SaveSlice(J2XY_cpu, J2XY_cuda, nX - 1, nY - 1, nZ, nZ / 2, "data/J2XYcXY_" + std::to_string(8 * NGRID) + "_.dat");
    SaveSlice(J2XZ_cpu, J2XZ_cuda, nX - 1, nY, nZ - 1, nZ / 2, "data/J2XZcXY_" + std::to_string(8 * NGRID) + "_.dat");
    SaveSlice(J2YZ_cpu, J2YZ_cuda, nX, nY - 1, nZ - 1, nZ / 2, "data/J2YZcXY_" + std::to_string(8 * NGRID) + "_.dat");
  }
  SaveSlice(Ux_cpu, Ux_cuda, nX + 1, nY, nZ, nZ / 2, "data/UxcXY_" + std::to_string(8 * NGRID) + "_.dat");
  SaveSlice(Vx_cpu, Vx_cuda, nX + 1, nY, nZ, nZ / 2, "data/VxcXY_" + std::to_string(8 * NGRID) + "_.dat");
  //SaveMatrix(tauYY_cpu, tauYY_cuda, nX, nY, "data/tauYYc_" + std::to_string(32 * NGRID) + "_.dat");
  //SaveMatrix(tauXYav_cpu, tauXYav_cuda, nX, nY, "data/tauXYavc_" + std::to_string(32 * NGRID) + "_.dat");
  //SaveMatrix(J2_cpu, J2_cuda, nX, nY, "data/J2c_" + std::to_string(32 * NGRID) + "_.dat");
  //SaveMatrix(Uy_cpu, Uy_cuda, nX, nY + 1, "data/Uyc_" + std::to_string(32 * NGRID) + "_.dat");

  /*const double tauXYmax = findMaxAbs(tauXZ_cpu, (nX - 1) * nY * (nZ - 1));
  std::cout << "tauXYmax = " << tauXYmax << "\n";*/

  //gpuErrchk(hipDeviceReset());
  const auto end = std::chrono::system_clock::now();
  int elapsed_sec = static_cast<int>(std::chrono::duration_cast<std::chrono::seconds>(end - start).count());
  printDuration(elapsed_sec);

  return 0.0;
}

void EffPlast3D::ComputeEffParams(const size_t step, const double loadStepValue, const std::array<double, 6>& loadType, const size_t nTimeSteps) {
  printStepInfo(step);

  PeffNonper[step].resize(nTimeSteps);
  PeffPer[step].resize(nTimeSteps);
  /*tauInfty[step].resize(nTimeSteps);*/
  dPhiNonper[step].resize(nTimeSteps);
  dPhiPer[step].resize(nTimeSteps);
  /*epsilon[step].resize(nTimeSteps);
  epsilonPer[step].resize(nTimeSteps);
  sigma[step].resize(nTimeSteps);
  sigmaPer[step].resize(nTimeSteps);*/

  double dUxdx = 0.0;
  double dUydy = 0.0;
  double dUzdz = 0.0;
  double dUxdy = 0.0;
  double dUxdz = 0.0;
  double dUydz = 0.0;

  if (step == 0) {
    curEffStrain = { 0.0 };
    memset(Ux_cpu, 0, (nX + 1) * nY * nZ * sizeof(double));
    memset(Uy_cpu, 0, nX * (nY + 1) * nZ * sizeof(double));
    memset(Uz_cpu, 0, nX * nY * (nZ + 1) * sizeof(double));
  }
  else { // additional loading
    gpuErrchk(hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(Uz_cpu, Uz_cuda, nX * nY * (nZ + 1) * sizeof(double), hipMemcpyDeviceToHost));
  }

  /* ACTION LOOP */
  for (int it = 0; it < nTimeSteps; it++) {
    std::cout << "Time step " << (it + 1) << " from " << nTimeSteps << "\n";
    log_file << "Time step " << (it + 1) << " from " << nTimeSteps << "\n";

    /*epsilon[step][it] = { 0.0 };
    epsilonPer[step][it] = { 0.0 };
    sigma[step][it] = { 0.0 };
    sigmaPer[step][it] = { 0.0 };*/

    dUxdx = loadStepValue * loadType[0] / static_cast<double>(nTimeSteps);
    dUydy = loadStepValue * loadType[1] / static_cast<double>(nTimeSteps);
    dUzdz = loadStepValue * loadType[2] / static_cast<double>(nTimeSteps);
    dUxdy = loadStepValue * loadType[3] / static_cast<double>(nTimeSteps);
    dUxdz = loadStepValue * loadType[4] / static_cast<double>(nTimeSteps);
    dUydz = loadStepValue * loadType[5] / static_cast<double>(nTimeSteps);
    //dUydx = dUxdy;

    curEffStrain[0] += dUxdx;
    curEffStrain[1] += dUydy;
    curEffStrain[2] += dUzdz;
    curEffStrain[3] += dUxdy;
    curEffStrain[4] += dUxdz;
    curEffStrain[5] += dUydz;
    //epsilon[step][it] = curEffStrain;

    std::cout << "Macro strain: (" << curEffStrain[0] << ", " << curEffStrain[1] << ", " << curEffStrain[2] << ", " << curEffStrain[3] << ", " << curEffStrain[4] << ", " << curEffStrain[5] << ")\n";
    log_file << "Macro strain: (" << curEffStrain[0] << ", " << curEffStrain[1] << ", " << curEffStrain[2] << ", " << curEffStrain[3] << ", " << curEffStrain[4] << ", " << curEffStrain[5] << ")\n";

    if (it > 0) {    // non-first time step
      gpuErrchk(hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * nZ * sizeof(double), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(Uz_cpu, Uz_cuda, nX * nY * (nZ + 1) * sizeof(double), hipMemcpyDeviceToHost));
    }

    //std::cout << "Ux = " << Ux_cpu[(3 * nY / 4) * (nX + 1) + 3 * nX / 4] << "\nUy = " << Uy_cpu[(3 * nY / 4) * nX + 3 * nX / 4] << "\n";

    for (int i = 0; i < nX + 1; i++) {
      for (int j = 0; j < nY; j++) {
        for (int k = 0; k < nZ; k++) {
          Ux_cpu[k * (nX + 1) * nY + j * (nX + 1) + i] += (-0.5 * dX * nX + dX * i) * dUxdx + (-0.5 * dY * (nY - 1) + dY * j) * dUxdy + (-0.5 * dZ * (nZ - 1) + dZ * k) * dUxdz;
        }
      }
    }
    gpuErrchk(hipMemcpy(Ux_cuda, Ux_cpu, (nX + 1) * nY * nZ * sizeof(double), hipMemcpyHostToDevice));
    for (int i = 0; i < nX; i++) {
      for (int j = 0; j < nY + 1; j++) {
        for (int k = 0; k < nZ; k++) {
          Uy_cpu[k * nX * (nY + 1) + j * nX + i] += (-0.5 * dY * nY + dY * j) * dUydy + (-0.5 * dZ * (nZ - 1) + dZ * k) * dUydz;
        }
      }
    }
    gpuErrchk(hipMemcpy(Uy_cuda, Uy_cpu, nX * (nY + 1) * nZ * sizeof(double), hipMemcpyHostToDevice));
    for (int i = 0; i < nX; i++) {
      for (int j = 0; j < nY; j++) {
        for (int k = 0; k < nZ + 1; k++) {
          Uz_cpu[k * nX * nY + j * nX + i] += (-0.5 * dZ * nZ + dZ * k) * dUzdz;
        }
      }
    }
    gpuErrchk(hipMemcpy(Uz_cuda, Uz_cpu, nX * nY * (nZ + 1) * sizeof(double), hipMemcpyHostToDevice));

    //std::cout << "dUxdx = " << dUxdx << "\ndUydy = " << dUydy << "\ndUxdy = " << dUxdy << "\n";
    //std::cout << "Ux = " << Ux_cpu[(3 * nY / 4) * (nX + 1) /*+ 3 * nX / 4*/] << "\nUy = " << Uy_cpu[(3 * nY / 4) * nX /*+ 3 * nX / 4*/] << "\n";

    double error = 0.0;

    /* ITERATION LOOP */
    for (int iter = 0; iter < NITER; iter++) {
      ComputeStress<<<grid, block>>>(Ux_cuda, Uy_cuda, Uz_cuda,
        K_cuda, G_cuda, P0_cuda, P_cuda,
        tauXX_cuda, tauYY_cuda, tauZZ_cuda,
        tauXY_cuda, tauXZ_cuda, tauYZ_cuda,
        /*tauXYav_cuda, J2_cuda, J2XY_cuda,*/ pa_cuda, nX, nY, nZ);
      gpuErrchk(hipDeviceSynchronize());
      
      if (NL > 1) {
        ComputeJ2<<<grid, block>>>(tauXX_cuda, tauYY_cuda, tauZZ_cuda,
          tauXY_cuda, tauXZ_cuda, tauYZ_cuda,
          tauXYav_cuda, tauXZav_cuda, tauYZav_cuda,
          J2_cuda, J2XY_cuda, J2XZ_cuda, J2YZ_cuda,
          nX, nY, nZ);
        gpuErrchk(hipDeviceSynchronize());
        ComputePlasticity<<<grid, block>>>(tauXX_cuda, tauYY_cuda, tauZZ_cuda,
          tauXY_cuda, tauXZ_cuda, tauYZ_cuda,
          tauXYav_cuda, tauXZav_cuda, tauYZav_cuda,
          J2_cuda, J2XY_cuda, J2XZ_cuda, J2YZ_cuda,
          nX, nY, nZ);
        gpuErrchk(hipDeviceSynchronize());
      }
      ComputeDisp<<<grid, block>>>(Ux_cuda, Uy_cuda, Uz_cuda,
        Vx_cuda, Vy_cuda, Vz_cuda, P_cuda,
        tauXX_cuda, tauYY_cuda, tauZZ_cuda,
        tauXY_cuda, tauXZ_cuda, tauYZ_cuda,
        pa_cuda, nX, nY, nZ);
      gpuErrchk(hipDeviceSynchronize());

      /*if (iter == 1000) {
      gpuErrchk(hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * sizeof(double), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * sizeof(double), hipMemcpyDeviceToHost));
      std::cout << "Ux1 = " << Ux_cpu[(3 * nY / 4) * (nX + 1) + 3 * nX / 4] << "\nUy1 = " << Uy_cpu[(3 * nY / 4) * nX + 3 * nX / 4] << "\n";
      }*/

      if ((iter + 1) % output_step == 0) {
        gpuErrchk(hipMemcpy(Vx_cpu, Vx_cuda, (nX + 1) * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Vy_cpu, Vy_cuda, nX * (nY + 1) * nZ * sizeof(double), hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(Vz_cpu, Vz_cuda, nX * nY * (nZ + 1) * sizeof(double), hipMemcpyDeviceToHost));

        error = (
          findMaxAbs(Vx_cpu, (nX + 1) * nY * nZ) / (dX * (nX - 1)) + 
          findMaxAbs(Vy_cpu, nX * (nY + 1) * nZ) / (dY * (nY - 1)) +
          findMaxAbs(Vz_cpu, nX * nY * (nZ + 1)) / (dZ * (nZ - 1))
        ) * dT /
          (std::max(std::abs(curEffStrain[0]), std::max(curEffStrain[1], curEffStrain[2])));
        //(std::abs(loadStepValue) * std::max(std::max(std::abs(loadType[0]), std::abs(loadType[1])), std::abs(loadType[2])));

        std::cout << "    Iteration " << iter + 1 << ": Error is " << error << std::endl;
        log_file << "    Iteration " << iter + 1 << ": Error is " << error << std::endl;

        if (error < EITER) {
          std::cout << "Number of iterations is " << iter + 1 << "\n\n";
          log_file << "Number of iterations is " << iter + 1 << "\n\n";
          break;
        }
        else if (iter >= NITER - 1) {
          std::cout << "WARNING: Maximum number of iterations reached!\nError is " << error << "\n\n";
          log_file << "WARNING: Maximum number of iterations reached!\nError is " << error << "\n\n";
        }
      }
    } // for(iter), iteration loop

    /* AVERAGING */
    gpuErrchk(hipMemcpy(P_cpu, P_cuda, nX * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
    /*gpuErrchk(hipMemcpy(tauXX_cpu, tauXX_cuda, nX * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tauYY_cpu, tauYY_cuda, nX * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tauZZ_cpu, tauZZ_cuda, nX * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tauXY_cpu, tauXY_cuda, (nX - 1) * (nY - 1) * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tauXZ_cpu, tauXZ_cuda, (nX - 1) * nY * (nZ - 1) * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(tauYZ_cpu, tauYZ_cuda, nX * (nY - 1) * (nZ - 1) * sizeof(double), hipMemcpyDeviceToHost));
    //gpuErrchk(hipMemcpy(tauXYav_cpu, tauXYav_cuda, nX * nY * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(J2_cpu, J2_cuda, nX * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));*/
    gpuErrchk(hipMemcpy(Ux_cpu, Ux_cuda, (nX + 1) * nY * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(Uy_cpu, Uy_cuda, nX * (nY + 1) * nZ * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(Uz_cpu, Uz_cuda, nX * nY * (nZ + 1) * sizeof(double), hipMemcpyDeviceToHost));

    PeffNonper[step][it] = getPeffNonper();
    PeffPer[step][it] = getPeffPer();

    std::cout << "    P / Y = " << PeffNonper[step][it] / Y << '\n';
    log_file << "    P / Y = " << PeffNonper[step][it] / Y << '\n';
    if (nPores > 2) {
      std::cout << "    Pper / Y = " << PeffPer[step][it] / Y << '\n';
      log_file << "    Pper / Y = " << PeffPer[step][it] / Y << '\n';
    }

    zeroingPoresDisp();
    /*SaveSlice(Ux_cpu, Ux_cuda, nX + 1, nY, nZ, nZ / 2, "data/UxcXY_" + std::to_string(8 * NGRID) + "_.dat");
    SaveSlice(Uy_cpu, Uy_cuda, nX, nY + 1, nZ, nZ / 2, "data/UycXY_" + std::to_string(8 * NGRID) + "_.dat");
    SaveSlice(Uz_cpu, Uz_cuda, nX, nY, nZ + 1, nZ / 2, "data/UzcXY_" + std::to_string(8 * NGRID) + "_.dat");*/
    calcPoreVolume();

    const double Phi0 = 3.1415926 * 4.0 * pow(rad * nPores, 3.0) / (3.0 * lX * lY * lZ);
    //std::cout << "    Phi0 = " << Phi0 << '\n';
    log_file << "    Phi0 = " << Phi0 << '\n';
    const double PhiNonper = 3.1415926 * 4.0 * poreVolume43Pi / (3.0 * lX * lY * lZ);
    //std::cout << "    PhiNonper = " << PhiNonper << '\n';
    log_file << "    PhiNonper = " << PhiNonper << '\n';
    const double PhiPer = nPores > 2 ? 
      3.1415926 * 4.0 * internalPoreVolume43Pi / (3.0 * lX * lY * lZ * pow(static_cast<double>(nPores - 2) / nPores, 3.0)) :
      0.0;
    //std::cout << "    PhiPer = " << PhiPer << '\n';
    log_file << "    PhiPer = " << PhiPer << '\n';


    dPhiNonper[step][it] = std::abs(PhiNonper - Phi0);
    std::cout << "    dPhiNonper = " << dPhiNonper[step][it] << '\n';
    log_file << "    dPhiNonper = " << dPhiNonper[step][it] << '\n';
    dPhiPer[step][it] = std::abs(PhiPer - Phi0);
    std::cout << "    dPhiPer = " << dPhiPer[step][it] << '\n';
    log_file << "    dPhiPer = " << dPhiPer[step][it] << '\n';
  } // for(it), action loop
}

void EffPlast3D::ReadParams(const std::string& filename) {
  std::ifstream pa_fil(filename, std::ios_base::binary);
  if (!pa_fil.is_open()) {
    throw std::runtime_error("ERROR:  Cannot open file " + filename + "!\n");
  }
  pa_fil.read((char*)pa_cpu, sizeof(double) * NPARS);
  gpuErrchk(hipMemcpy(pa_cuda, pa_cpu, NPARS * sizeof(double), hipMemcpyHostToDevice));
}
void EffPlast3D::SetMaterials() {
  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      for (int k = 0; k < nZ; k++) {
        K_cpu[k * nX * nY + j * nX + i] = K0;
        G_cpu[k * nX * nY + j * nX + i] = G0;
        const double x = -0.5 * dX * (nX - 1) + dX * i;
        const double y = -0.5 * dY * (nY - 1) + dY * j;
        const double z = -0.5 * dZ * (nZ - 1) + dZ * k;
        const double Lx = dX * (nX - 1);
        const double Ly = dY * (nY - 1);
        const double Lz = dZ * (nZ - 1);
        for (int a = 0; a < nPores; a++) {
          for (int b = 0; b < nPores; b++) {
            for (int c = 0; c < nPores; c++) {
              if (sqrt(
                (x - 0.5 * Lx * (1.0 - 1.0 / nPores) + (Lx / nPores) * a) * (x - 0.5 * Lx * (1.0 - 1.0 / nPores) + (Lx / nPores) * a) +
                (y - 0.5 * Ly * (1.0 - 1.0 / nPores) + (Ly / nPores) * b) * (y - 0.5 * Ly * (1.0 - 1.0 / nPores) + (Ly / nPores) * b) +
                (z - 0.5 * Lz * (1.0 - 1.0 / nPores) + (Lz / nPores) * c) * (z - 0.5 * Lz * (1.0 - 1.0 / nPores) + (Lz / nPores) * c)
              ) < rad) {
                K_cpu[k * nX * nY + j * nX + i] = 0.01 * K0;
                G_cpu[k * nX * nY + j * nX + i] = 0.01 * G0;
                //empty_spaces.emplace(i, j);
              }
            } // for(c)
          } // for(b)
        } // for(a)
      } // for(k)
    } // for(j)
  } // for(i)
  
  gpuErrchk(hipMemcpy(K_cuda, K_cpu, nX * nY * nZ * sizeof(double), hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(G_cuda, G_cpu, nX * nY * nZ * sizeof(double), hipMemcpyHostToDevice));
}
void EffPlast3D::SetInitPressure(const double coh) {
  const double P0 = 0.0; //1.0 * coh;

  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      for (int k = 0; k < nZ; k++) {
        P0_cpu[j * nX + i] = 0.0;
        if (sqrt(
          (-0.5 * dX * (nX - 1) + dX * i) * (-0.5 * dX * (nX - 1) + dX * i) + 
          (-0.5 * dY * (nY - 1) + dY * j) * (-0.5 * dY * (nY - 1) + dY * j) +
          (-0.5 * dZ * (nZ - 1) + dZ * k) * (-0.5 * dZ * (nZ - 1) + dZ * k)
        ) < rad) {
          P0_cpu[k * nX * nY + j * nX + i] = P0;
        }
      } // for(k)
    } // for(j)
  } // for(i)

  gpuErrchk(hipMemcpy(P0_cuda, P0_cpu, nX * nY * nZ * sizeof(double), hipMemcpyHostToDevice));
}

void EffPlast3D::SetTensorZero(double** A_cpu, double** A_cuda, const int m, const int n, const int o) {
  *A_cpu = new double[m * n * o];
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < n; j++) {
      for (int k = 0; k < o; k++) {
        (*A_cpu)[k * m * n + j * m + i] = 0.0;
      }
    }
  }
  gpuErrchk(hipMalloc(A_cuda, m * n * o * sizeof(double)));
  gpuErrchk(hipMemcpy(*A_cuda, *A_cpu, m * n * o * sizeof(double), hipMemcpyHostToDevice));
}
void EffPlast3D::SaveSlice(double* const A_cpu, const double* const A_cuda, const int m, const int n, const int o, const int k, const std::string& filename) {
  gpuErrchk(hipMemcpy(A_cpu, A_cuda, m * n * o * sizeof(double), hipMemcpyDeviceToHost));
  std::ofstream A_filw(filename, std::ios_base::binary);
  A_filw.write((char*)A_cpu + sizeof(double) * m * n * k, sizeof(double) * m * n);
}

double EffPlast3D::findMaxAbs(const double* const arr, const int size) {
  double max_el = 0.0;
  for (int i = 0; i < size; i++) {
    if (std::abs(arr[i]) > std::abs(max_el)) {
      max_el = std::abs(arr[i]);
    }
  }
  return max_el;
}
double EffPlast3D::findMaxAbs(const std::vector<double>& vec) {
  double max_el = 0.0;
  for (auto i : vec) {
    if (std::abs(i) > std::abs(max_el)) {
      max_el = i;
    }
  }
  return max_el;
}
void EffPlast3D::zeroingPoresDisp() {
  // set zero Ux in the pores
  for (int i = 0; i < nX + 1; i++) {
    for (int j = 0; j < nY; j++) {
      for (int k = 0; k < nZ; k++) {
        const double x = -0.5 * dX * nX + dX * i;
        const double y = -0.5 * dY * (nY - 1) + dY * j;
        const double z = -0.5 * dZ * (nZ - 1) + dZ * k;
        for (int a = 0; a < nPores; a++) {
          for (int b = 0; b < nPores; b++) {
            for (int c = 0; c < nPores; c++) {
              if (sqrt(
                (x - 0.5 * lX * (1.0 - 1.0 / nPores) + (lX / nPores) * a) * (x - 0.5 * lX * (1.0 - 1.0 / nPores) + (lX / nPores) * a) +
                (y - 0.5 * lY * (1.0 - 1.0 / nPores) + (lY / nPores) * b) * (y - 0.5 * lY * (1.0 - 1.0 / nPores) + (lY / nPores) * b) +
                (z - 0.5 * lZ * (1.0 - 1.0 / nPores) + (lZ / nPores) * c) * (z - 0.5 * lZ * (1.0 - 1.0 / nPores) + (lZ / nPores) * c)
              ) < rad)
              {
                Ux_cpu[k * (nX + 1) * nY + j * (nX + 1) + i] = 0.0;
              }
            } // for(c)
          } // for(b)
        } // for(a)
      } // for(k)
    } // for(j)
  }
  // set zero Uy in the pores
  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY + 1; j++) {
      for (int k = 0; k < nZ; k++) {
        const double x = -0.5 * dX * (nX - 1) + dX * i;
        const double y = -0.5 * dY * nY + dY * j;
        const double z = -0.5 * dZ * (nZ - 1) + dZ * k;
        for (int a = 0; a < nPores; a++) {
          for (int b = 0; b < nPores; b++) {
            for (int c = 0; c < nPores; c++) {
              if (sqrt(
                (x - 0.5 * lX * (1.0 - 1.0 / nPores) + (lX / nPores) * a) * (x - 0.5 * lX * (1.0 - 1.0 / nPores) + (lX / nPores) * a) +
                (y - 0.5 * lY * (1.0 - 1.0 / nPores) + (lY / nPores) * b) * (y - 0.5 * lY * (1.0 - 1.0 / nPores) + (lY / nPores) * b) +
                (z - 0.5 * lZ * (1.0 - 1.0 / nPores) + (lZ / nPores) * c) * (z - 0.5 * lZ * (1.0 - 1.0 / nPores) + (lZ / nPores) * c)
              ) < rad)
              {
                Uy_cpu[k * nX * (nY + 1) + j * nX + i] = 0.0;
              }
            } // for(c)
          } // for(b)
        } // for(a)
      } // for(k)
    } // for(j)
  }
  // set zero Uz in the pores
  for (int i = 0; i < nX; i++) {
    for (int j = 0; j < nY; j++) {
      for (int k = 0; k < nZ + 1; k++) {
        const double x = -0.5 * dX * (nX - 1) + dX * i;
        const double y = -0.5 * dY * (nY - 1) + dY * j;
        const double z = -0.5 * dZ * nZ + dZ * k;
        for (int a = 0; a < nPores; a++) {
          for (int b = 0; b < nPores; b++) {
            for (int c = 0; c < nPores; c++) {
              if (sqrt(
                (x - 0.5 * lX * (1.0 - 1.0 / nPores) + (lX / nPores) * a) * (x - 0.5 * lX * (1.0 - 1.0 / nPores) + (lX / nPores) * a) +
                (y - 0.5 * lY * (1.0 - 1.0 / nPores) + (lY / nPores) * b) * (y - 0.5 * lY * (1.0 - 1.0 / nPores) + (lY / nPores) * b) +
                (z - 0.5 * lZ * (1.0 - 1.0 / nPores) + (lZ / nPores) * c) * (z - 0.5 * lZ * (1.0 - 1.0 / nPores) + (lZ / nPores) * c)
              ) < rad)
              {
                Uz_cpu[k * nX * nY + j * nX + i] = 0.0;
              }
            } // for(c)
          } // for(b)
        } // for(a)
      } // for(k)
    } // for(j)
  }
}
void EffPlast3D::calcPoreVolume() const {
  poreVolume43Pi = 0.0;
  internalPoreVolume43Pi = 0.0;
  for (int a = 0; a < nPores; a++) {
    for (int b = 0; b < nPores; b++) {
      for (int c = 0; c < nPores; c++) {
        const double cxdX = 0.5 * (nX - 1) * (1.0 - 1.0 / nPores) - (static_cast<double>(nX - 1) / nPores) * a; // cx / dX
        const double cydY = 0.5 * (nY - 1) * (1.0 - 1.0 / nPores) - (static_cast<double>(nY - 1) / nPores) * b; // cy / dY
        const double czdZ = 0.5 * (nZ - 1) * (1.0 - 1.0 / nPores) - (static_cast<double>(nZ - 1) / nPores) * c; // cz / dZ

        const size_t cxIdx = static_cast<size_t>(cxdX + 0.5 * (nX - 1));
        const size_t cyIdx = static_cast<size_t>(cydY + 0.5 * (nY - 1));
        const size_t czIdx = static_cast<size_t>(czdZ + 0.5 * (nZ - 1));

        // horizontal displacements
        // left point of a pore        
        size_t rxIdx = static_cast<size_t>(cxdX - rad / dX + 0.5 * nX);
        std::vector<double> dispXleft(5);
        //std::cout << "dispXleft:\n";
        if (rxIdx < 1) {
          throw std::out_of_range("Error in calcPoreVolume! Grid is too small or pores are too big!\n");
        }
        for (int i = 0; i < 5; i++) {
          dispXleft[i] = Ux_cpu[czIdx * (nX + 1) * nY + cyIdx * (nX + 1) + rxIdx - 1 + i];
          //std::cout << "j = " << cyIdx << " i = " << rxIdx - 1 + i << "\n";
          //std::cout << dispXleft[i] << "\n";
        }
        // right point of a pore
        rxIdx = static_cast<size_t>(cxdX + rad / dX + 0.5 * nX);
        std::vector<double> dispXright(5);
        //std::cout << "dispXright:\n";
        if (rxIdx > nX - 2) {
          throw std::out_of_range("Error in calcPoreVolume! Grid is too small or pores are too big!\n");
        }
        for (int i = 0; i < 5; i++) {
          dispXright[i] = Ux_cpu[czIdx * (nX + 1) * nY + cyIdx * (nX + 1) + rxIdx - 2 + i];
          //std::cout << dispXright[i] << "\n";
        }

        // depth displacements
        // near point of a pore
        size_t ryIdx = static_cast<size_t>(cydY - rad / dY + 0.5 * nY);
        std::vector<double> dispYnear(5);
        //std::cout << "dispYnear:\n";
        if (ryIdx < 1) {
          throw std::out_of_range("Error in calcPoreVolume! Grid is too small or pores are too big!\n");
        }
        for (int j = 0; j < 5; j++) {
          dispYnear[j] = Uy_cpu[czIdx * nX * (nY + 1) + (ryIdx - 1 + j) * nX + cxIdx];
          //std::cout << dispYnear[j] << "\n";
        }
        // far point of a hole
        ryIdx = static_cast<size_t>(cydY + rad / dY + 0.5 * nY);
        std::vector<double> dispYfar(5);
        //std::cout << "dispYfar:\n";
        if (ryIdx > nY - 2) {
          throw std::out_of_range("Error in calcPoreVolume! Grid is too small or pores are too big!\n");
        }
        for (int j = 0; j < 5; j++) {
          dispYfar[j] = Uy_cpu[czIdx * nX * (nY + 1) + (ryIdx - 2 + j) * nX + cxIdx];
          //std::cout << dispYfar[j] << "\n";
        }

        // vertical displacements
        // bottom point of a hole
        size_t rzIdx = static_cast<size_t>(czdZ - rad / dZ + 0.5 * nZ);
        std::vector<double> dispZbottom(5);
        //std::cout << "dispZbottom:\n";
        if (rzIdx < 1) {
          throw std::out_of_range("Error in calcPoreVolume! Grid is too small or pores are too big!\n");
        }
        for (int k = 0; k < 5; k++) {
          dispZbottom[k] = Uz_cpu[(rzIdx - 1 + k) * nX * nY + cyIdx * nX + cxIdx];
          //std::cout << dispZbottom[k] << "\n";
        }
        // top point of a hole
        rzIdx = static_cast<size_t>(czdZ + rad / dZ + 0.5 * nZ);
        std::vector<double> dispZtop(5);
        //std::cout << "dispYtop\n";
        if (rzIdx > nZ - 2) {
          throw std::out_of_range("Error in calcPoreVolume! Grid is too small or pores are too big!\n");
        }
        for (int k = 0; k < 5; k++) {
          dispZtop[k] = Uz_cpu[(rzIdx - 2 + k) * nX * nY + cyIdx * nX + cxIdx];
          //std::cout << dispZtop[k] << "\n";
        }

        //std::cout << "dRxLeft = " << FindMaxAbs(dispXleft) << ", dRxRight = " << FindMaxAbs(dispXright) << "\n";
        const double dRx = -0.5 * (findMaxAbs(dispXleft) - findMaxAbs(dispXright));
        const double dRy = -0.5 * (findMaxAbs(dispYnear) - findMaxAbs(dispYfar));
        const double dRz = -0.5 * (findMaxAbs(dispZbottom) - findMaxAbs(dispZtop));
        //std::cout << "dRx = " << dRx << ", dRy = " << dRy << "\n";

        poreVolume43Pi += (rad + dRx) * (rad + dRy) * (rad + dRz);
        //std::cout << poreVolume43Pi << "\n";
        if (a > 0 && b > 0 && c > 0 && a < nPores - 1 && b < nPores - 1 && c < nPores - 1) {
          internalPoreVolume43Pi += (rad + dRx) * (rad + dRy) * (rad + dRz);
        }
      } // for(c)
    } // for(b)
  } // for(a)
}

/* AVERAGING */
double EffPlast3D::getPeffNonper() const {
  if (nX <= 2 && nY <= 2 && nZ <= 2) {
    throw std::runtime_error("Error in getPeffNonper! The grid is too small!\n");
  }
  double PeffX{0.0}, PeffY{0.0}, PeffZ{0.0};
  for (int j = 1; j < nY - 1; j++) {
    for (int k = 1; k < nZ - 1; k++) {
      PeffX += P_cpu[k * nX * nY + j * nX + 0];
      PeffX += P_cpu[k * nX * nY + j * nX + nX - 1];
    }
  }
  PeffX /= 2.0 * (nY - 2) * (nZ - 2);
  for (int i = 1; i < nX - 1; i++) {
    for (int k = 1; k < nZ - 1; k++) {
      PeffY += P_cpu[k * nX * nY + 0 * nX + i];
      PeffY += P_cpu[k * nX * nY + (nY - 1) * nX + i];
    }
  }
  PeffY /= 2.0 * (nX - 2) * (nZ - 2);
  for (int i = 1; i < nX - 1; i++) {
    for (int j = 1; j < nY - 1; j++) {
      PeffZ += P_cpu[0 * nX * nY + j * nX + i];
      PeffZ += P_cpu[(nZ - 1) * nX * nY + j * nX + i];
    }
  }
  PeffZ /= 2.0 * (nX - 2) * (nY - 2);
  return (PeffX + PeffY + PeffZ) / 3.0;
}
double EffPlast3D::getPeffPer() const {
  if (nPores <= 2) {
    return 0.0;
  }
  if (nX < nPores || nY < nPores || nZ < nPores) {
    throw std::runtime_error("Error in getPeffPer! The grid is too small!\n");
  }
  double PeffX{0.0}, PeffY{0.0}, PeffZ{0.0};
  for (int j = nY / nPores; j < nY * (nPores - 1) / nPores; j++) {
    for (int k = nZ / nPores; k < nZ * (nPores - 1) / nPores; k++) {
      PeffX += P_cpu[k * nX * nY + j * nX + nX / nPores];
      PeffX += P_cpu[k * nX * nY + j * nX + nX  * (nPores - 1) / nPores];
    }
  }
  PeffX /= 2.0 * (nY - 2) * (nZ - 2) * (nPores - 2) * (nPores - 2) / nPores / nPores;
  for (int i = nX / nPores; i < nX * (nPores - 1) / nPores; i++) {
    for (int k = nZ / nPores; k < nZ * (nPores - 1) / nPores; k++) {
      PeffY += P_cpu[k * nX * nY + nY / nPores * nX + i];
      PeffY += P_cpu[k * nX * nY + nY * (nPores - 1) / nPores * nX + i];
    }
  }
  PeffY /= 2.0 * (nX - 2) * (nZ - 2) * (nPores - 2) * (nPores - 2) / nPores / nPores;
  for (int i = nX / nPores; i < nX * (nPores - 1) / nPores; i++) {
    for (int j = nY / nPores; j < nY * (nPores - 1) / nPores; j++) {
      PeffZ += P_cpu[nZ / nPores * nX * nY + j * nX + i];
      PeffZ += P_cpu[nZ * (nPores - 1) / nPores * nX * nY + j * nX + i];
    }
  }
  PeffZ /= 2.0 * (nX - 2) * (nY - 2) * (nPores - 2) * (nPores - 2) / nPores / nPores;
  return (PeffX + PeffY + PeffZ) / 3.0;
}

/* CONSOLE AND LOG FILE OUTPUT */
void EffPlast3D::printStepInfo(const size_t step) {
  std::cout << "\nLOAD STEP " << step + 1 << " FROM " << NL << ": ";
  log_file << "\nLOAD STEP " << step + 1 << " FROM " << NL << ": ";
  switch (step) {
  case 0:
    std::cout << "PRELOADING\n";
    log_file << "PRELOADING\n";
    break;
  case 1:
    std::cout << "SMALL HYDROSTATIC INCREMENT\n";
    log_file << "SMALL HYDROSTATIC INCREMENT\n";
    break;
  case 2:
    std::cout << "SMALL DEVIATORIC INCREMENT\n";
    log_file << "SMALL DEVIATORIC INCREMENT\n";
    break;
  default:
    throw std::invalid_argument("ERROR:  Wrong step index!\n");
  }
  std::cout << "Porosity is " << porosity * 100 << "%\n";
  log_file << "Porosity is " << porosity * 100 << "%\n";
  std::cout << "Grid resolution is " << nX << "x" << nY << "x" << nZ << "\n\n";
  log_file << "Grid resolution is " << nX << "x" << nY << "x" << nZ << "\n\n";
}
void EffPlast3D::printCalculationType() {
  switch (NL) {
  case 1:
    std::cout << "\nPURE ELASTIC CALCULATION\nESTIMATION OF THE EFFECTIVE BULK MODULI\n";
    log_file << "\nPURE ELASTIC CALCULATION\nESTIMATION OF THE EFFECTIVE BULK MODULI\n";
    break;
  case 2:
    std::cout << "\nELASTOPLASTIC CALCULATION\nESTIMATION OF THE EFFECTIVE BULK MODULI\n";
    log_file << "\nELASTOPLASTIC CALCULATION\nESTIMATION OF THE EFFECTIVE BULK MODULI\n";
    break;
  /*case 3:
    std::cout << "\nELASTOPLASTIC CALCULATION\nESTIMATION OF THE EFFECTIVE BULK MODULI AND THE EFFECTIVE SHEAR MODULUS\n";
    log_file << "\nELASTOPLASTIC CALCULATION\nESTIMATION OF THE EFFECTIVE BULK MODULI AND THE EFFECTIVE SHEAR MODULUS\n";
    break;*/
  default:
    throw std::invalid_argument("ERROR:  Wrong number of loads!\n");
  }
}
void EffPlast3D::printDuration(int elapsed_sec) {
  if (elapsed_sec < 60) {
    std::cout << "\nCalculation time is " << elapsed_sec << " sec\n";
    log_file << "\nCalculation time is " << elapsed_sec << " sec\n\n\n";
  }
  else {
    int elapsed_min = elapsed_sec / 60;
    elapsed_sec = elapsed_sec % 60;
    if (elapsed_min < 60) {
      std::cout << "\nCalculation time is " << elapsed_min << " min " << elapsed_sec << " sec\n";
      log_file << "\nCalculation time is " << elapsed_min << " min " << elapsed_sec << " sec\n\n\n";
    }
    else {
      int elapsed_hour = elapsed_min / 60;
      elapsed_min = elapsed_min % 60;
      if (elapsed_hour < 24) {
        std::cout << "\nCalculation time is " << elapsed_hour << " hours " << elapsed_min << " min " << elapsed_sec << " sec\n";
        log_file << "\nCalculation time is " << elapsed_hour << " hours " << elapsed_min << " min " << elapsed_sec << " sec\n\n\n";
      }
      else {
        const int elapsed_day = elapsed_hour / 24;
        elapsed_hour = elapsed_hour % 24;
        if (elapsed_day < 7) {
          std::cout << "\nCalculation time is " << elapsed_day << " days " << elapsed_hour << " hours " << elapsed_min << " min " << elapsed_sec << " sec\n";
          log_file << "\nCalculation time is " << elapsed_day << " days " << elapsed_hour << " hours " << elapsed_min << " min " << elapsed_sec << " sec\n\n\n";
        }
        else {
          std::cout << "\nCalculation time is " << elapsed_day / 7 << " weeks " << elapsed_day % 7 << " days " << elapsed_hour << " hours " << elapsed_min << " min " << elapsed_sec << " sec\n";
          log_file << "\nCalculation time is " << elapsed_day / 7 << " weeks " << elapsed_day % 7 << " days " << elapsed_hour << " hours " << elapsed_min << " min " << elapsed_sec << " sec\n\n\n";
        }
      }
    }
  }
}

/* FINAL EFFECTIVE MODULI CALCULATION */
void EffPlast3D::calcBulkModuli_PureElast() {
  eff_moduli_num_nonper.Kphi = getKphiNonper_PureElast();
  std::cout << "    ==============\n\n" << "KphiNonper = " << eff_moduli_num_nonper.Kphi << std::endl;
  log_file << "    ==============\n\n" << "KphiNonper = " << eff_moduli_num_nonper.Kphi << std::endl;

  eff_moduli_num_per.Kphi = getKphiPer_PureElast();
  std::cout << "KphiPer = " << eff_moduli_num_per.Kphi << std::endl;
  log_file << "KphiPer = " << eff_moduli_num_per.Kphi << std::endl;
}
void EffPlast3D::calcBulkModuli_ElastPlast() {
  eff_moduli_num_nonper.Kphi = getKphiNonper_ElastPlast();
  std::cout << "    ==============\n\n" << "KphiNonper = " << eff_moduli_num_nonper.Kphi << std::endl;
  log_file << "    ==============\n\n" << "KphiNonper = " << eff_moduli_num_nonper.Kphi << std::endl;

  eff_moduli_num_per.Kphi = getKphiPer_ElastPlast();
  std::cout << "KphiPer = " << eff_moduli_num_per.Kphi << std::endl;
  log_file << "KphiPer = " << eff_moduli_num_per.Kphi << std::endl;
}
// bulk moduli in the pure elastic case
double EffPlast3D::getKphiNonper_PureElast() {
  const double Pinc = PeffNonper[0][nTimeSteps_ - 1];
  const double phiInc = dPhiNonper[0][nTimeSteps_ - 1];
  return Pinc / phiInc;
}
double EffPlast3D::getKphiPer_PureElast() {
  const double Pinc = PeffPer[0][nTimeSteps_ - 1];
  const double phiInc = dPhiPer[0][nTimeSteps_ - 1];
  return Pinc / phiInc;
}
double EffPlast3D::getKphiNonper_ElastPlast() {
  const double Pinc = PeffNonper[1][0] - PeffNonper[0][nTimeSteps_ - 1];
  const double phiInc = dPhiNonper[1][0] - dPhiNonper[0][nTimeSteps_ - 1];
  return Pinc / phiInc;
}
double EffPlast3D::getKphiPer_ElastPlast() {
  const double Pinc = PeffPer[1][0] - PeffPer[0][nTimeSteps_ - 1];
  const double phiInc = dPhiPer[1][0] - dPhiPer[0][nTimeSteps_ - 1];
  return Pinc / phiInc;
}

EffPlast3D::EffPlast3D() {
  block.x = 8;
  block.y = 8;
  block.z = 8;
  grid.x = NGRID;
  grid.y = NGRID;
  grid.z = NGRID;

  nX = block.x * grid.x;
  nY = block.y * grid.y;
  nZ = block.z * grid.z;

  gpuErrchk(hipSetDevice(DEVICE_IDX));
  //gpuErrchk(hipDeviceReset());
  //gpuErrchk(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

  /* PARAMETERS */
  pa_cpu = new double[NPARS];
  gpuErrchk(hipMalloc(&pa_cuda, NPARS * sizeof(double)));
  ReadParams("data/pa.dat");

  dX = pa_cpu[0];
  dY = pa_cpu[1];
  dZ = pa_cpu[2];
  dT = pa_cpu[3];
  K0 = pa_cpu[4];
  G0 = pa_cpu[5];
  E0 = 9.0 * K0 * G0 / (3.0 * K0 + G0);
  nu0 = (1.5 * K0 - G0) / (3.0 * K0 + G0);
  //std::cout << "E = " << E0 << ", nu = " << nu0 << "\n";
  rad = pa_cpu[11];
  Y = pa_cpu[10] / sqrt(3.0);
  nPores = pa_cpu[12];

  hipMemcpyToSymbol(HIP_SYMBOL(coh_cuda), &pa_cpu[10], sizeof(double));

  /* SPACE ARRAYS */
  // materials
  K_cpu = new double[nX * nY * nZ];
  G_cpu = new double[nX * nY * nZ];
  gpuErrchk(hipMalloc(&K_cuda, nX * nY * nZ * sizeof(double)));
  gpuErrchk(hipMalloc(&G_cuda, nX * nY * nZ * sizeof(double)));
  SetMaterials();

  // stress
  P0_cpu = new double[nX * nY * nZ];
  gpuErrchk(hipMalloc(&P0_cuda, nX * nY * nZ * sizeof(double)));
  SetInitPressure(pa_cpu[10]);

  SetTensorZero(&P_cpu, &P_cuda, nX, nY, nZ);
  SetTensorZero(&tauXX_cpu, &tauXX_cuda, nX, nY, nZ);
  SetTensorZero(&tauYY_cpu, &tauYY_cuda, nX, nY, nZ);
  SetTensorZero(&tauZZ_cpu, &tauZZ_cuda, nX, nY, nZ);
  SetTensorZero(&tauXY_cpu, &tauXY_cuda, nX - 1, nY - 1, nZ);
  SetTensorZero(&tauXZ_cpu, &tauXZ_cuda, nX - 1, nY, nZ - 1);
  SetTensorZero(&tauYZ_cpu, &tauYZ_cuda, nX, nY - 1, nZ - 1);
  if (NL > 1) {
    SetTensorZero(&tauXYav_cpu, &tauXYav_cuda, nX, nY, nZ);
    SetTensorZero(&tauXZav_cpu, &tauXZav_cuda, nX, nY, nZ);
    SetTensorZero(&tauYZav_cpu, &tauYZav_cuda, nX, nY, nZ);

    // plasticity
    SetTensorZero(&J2_cpu, &J2_cuda, nX, nY, nZ);
    SetTensorZero(&J2XY_cpu, &J2XY_cuda, nX - 1, nY - 1, nZ);
    SetTensorZero(&J2XZ_cpu, &J2XZ_cuda, nX - 1, nY, nZ - 1);
    SetTensorZero(&J2YZ_cpu, &J2YZ_cuda, nX, nY - 1, nZ - 1);
  }

  // displacement
  SetTensorZero(&Ux_cpu, &Ux_cuda, nX + 1, nY, nZ);
  SetTensorZero(&Uy_cpu, &Uy_cuda, nX, nY + 1, nZ);
  SetTensorZero(&Uz_cpu, &Uz_cuda, nX, nY, nZ + 1);

  // velocity
  SetTensorZero(&Vx_cpu, &Vx_cuda, nX + 1, nY, nZ);
  SetTensorZero(&Vy_cpu, &Vy_cuda, nX, nY + 1, nZ);
  SetTensorZero(&Vz_cpu, &Vz_cuda, nX, nY, nZ + 1);

  /* UTILITIES */
  log_file.open("EffPlast3D.log", std::ios_base::app);
  output_step = 1000;
  lX = (nX - 1) * dX;
  lY = (nY - 1) * dY;
  lZ = (nZ - 1) * dZ;
  porosity = (4.0 / 3.0) * 3.1415926 * pow(rad * nPores, 3.0) / (lX * lY * lZ);
}
EffPlast3D::~EffPlast3D() {
  // parameters
  delete[] pa_cpu;
  gpuErrchk(hipFree(pa_cuda));

  // materials
  delete[] K_cpu;
  delete[] G_cpu;
  gpuErrchk(hipFree(K_cuda));
  gpuErrchk(hipFree(G_cuda));

  // stress
  delete[] P0_cpu;
  delete[] P_cpu;
  delete[] tauXX_cpu;
  delete[] tauYY_cpu;
  delete[] tauZZ_cpu;
  delete[] tauXY_cpu;
  delete[] tauXZ_cpu;
  delete[] tauYZ_cpu;
  //delete[] tauXYav_cpu;
  gpuErrchk(hipFree(P0_cuda));
  gpuErrchk(hipFree(P_cuda));
  gpuErrchk(hipFree(tauXX_cuda));
  gpuErrchk(hipFree(tauYY_cuda));
  gpuErrchk(hipFree(tauZZ_cuda));
  gpuErrchk(hipFree(tauXY_cuda));
  gpuErrchk(hipFree(tauXZ_cuda));
  gpuErrchk(hipFree(tauYZ_cuda));
  //gpuErrchk(hipFree(tauXYav_cuda));

  // plasticity
  /*delete[] J2_cpu;
  delete[] J2XY_cpu;
  gpuErrchk(hipFree(J2_cuda));
  gpuErrchk(hipFree(J2XY_cuda));*/

  // displacement
  delete[] Ux_cpu;
  delete[] Uy_cpu;
  delete[] Uz_cpu;
  gpuErrchk(hipFree(Ux_cuda));
  gpuErrchk(hipFree(Uy_cuda));
  gpuErrchk(hipFree(Uz_cuda));

  // velocity
  delete[] Vx_cpu;
  delete[] Vy_cpu;
  delete[] Vz_cpu;
  gpuErrchk(hipFree(Vx_cuda));
  gpuErrchk(hipFree(Vy_cuda));
  gpuErrchk(hipFree(Vz_cuda));

  // log
  log_file.close();
}